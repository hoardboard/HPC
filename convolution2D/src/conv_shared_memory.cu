#include "hip/hip_runtime.h"
#include "lodepng.cpp"
#include <iostream>
#include <string>
#include <sstream>

#define CUDA_CALL(F) if( (F) != hipSuccess ) \
{printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
    __FILE__,__LINE__); exit(-1);}
#define CUDA_CHECK() if( (hipPeekAtLastError()) != hipSuccess ) \
{printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
    __FILE__,__LINE__-1); exit(-1);}

using namespace std;

#define THPB 32
#define MASK_SIZE 3

__constant__ char g_filter[MASK_SIZE * MASK_SIZE];

__global__ void conv_kernel(unsigned char *image, unsigned char *ans,
                              int width, int height) {
  __shared__ int s_data[THPB + MASK_SIZE - 1][THPB + MASK_SIZE - 1];

  // Load data to shared memory
  const int radius = MASK_SIZE / 2;
  int dest = threadIdx.y * THPB + threadIdx.x,
      destY = dest / (THPB + MASK_SIZE - 1), destX = dest % (THPB + MASK_SIZE - 1),
      srcY = blockIdx.y * THPB + destY - radius,
      srcX = blockIdx.x * THPB + destX - radius,
      src = srcY * width + srcX;
  if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
    s_data[destY][destX] = image[src];
  else
    s_data[destY][destX] = 0;

  dest = threadIdx.y * THPB + threadIdx.x + THPB * THPB;
  destY = dest / (THPB + MASK_SIZE - 1), destX = dest % (THPB + MASK_SIZE - 1);
  srcY = blockIdx.y * THPB + destY - radius;
  srcX = blockIdx.x * THPB + destX - radius;
  src = srcY * width + srcX;
  if (destY < THPB + MASK_SIZE - 1) {
    if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
      s_data[destY][destX] = image[src];
    else
      s_data[destY][destX] = 0;
  }
  __syncthreads();



  int x = blockIdx.y * blockDim.y + threadIdx.y;
  int y = blockIdx.x * blockDim.x + threadIdx.x;

  if (x > height || y > width)
    return;

  int cur = 0, nx, ny;
  for (int i = 0; i < MASK_SIZE; ++i) {
    for (int j = 0; j < MASK_SIZE; ++j) {
      nx = threadIdx.y + i;
      ny = threadIdx.x + j;
      if (nx >= 0 && nx < height && ny >= 0 && ny < width) {
        cur += s_data[nx][ny] * g_filter[i * MASK_SIZE + j];
      }
    }
  }
  ans[x * width + y] = min(255, max(0, cur));

  __syncthreads();
}

double sequential(unsigned char *image, unsigned char *ans,
    int width, int height, char *filter, int f_size) {
  int dx[] = {-1, -1, -1, 0, 0, 0, 1, 1, 1};
  int dy[] = {-1, 0, 1, -1, 0, 1, -1, 0, 1};

  f_size = f_size * f_size;
  clock_t start = clock();
  for (int i = 0; i < height; ++i) {
    for (int j = 0; j < width; ++j) {
      int cur = 0;
      for (int k = 0; k < f_size; ++k) {
        int x = i + dx[k];
        int y = j + dy[k];
        if (x >= 0 && x < height && y >= 0 && y < width) {
          cur += filter[k] * image[x * width + y];
        }
      }
      ans[i * width + j] = min(255, max(0, cur));
    }
  }
  return (clock() - start) / (double) CLOCKS_PER_SEC;
}


double global_memory(unsigned char *image, unsigned char *ans,
    int width, int height, char *filter, int f_size) {
  clock_t start = clock();
  unsigned char *d_image, *d_ans;
  CUDA_CALL(hipMalloc(&d_image, width * height * sizeof(unsigned char)));
  CUDA_CALL(hipMalloc(&d_ans, width * height * sizeof(unsigned char)));

  CUDA_CALL(hipMemcpy(d_image, image, width * height * sizeof(unsigned char), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(g_filter), filter, f_size * f_size * sizeof(char)));

  dim3 dim_grid((width + THPB - 1) / THPB, (height + THPB - 1) / THPB, 1);
  dim3 dim_block(THPB, THPB, 1);

  conv_kernel<<< dim_grid, dim_block >>> (d_image, d_ans, width, height);
  CUDA_CHECK();

  CUDA_CALL(hipMemcpy(ans, d_ans, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));

  CUDA_CALL(hipFree(d_image));
  CUDA_CALL(hipFree(d_ans));
  return (clock() - start) / (double) CLOCKS_PER_SEC;
}

double tiled(unsigned char *image, int width, int height) {
  clock_t start = clock();

  return (clock() - start) / (double) CLOCKS_PER_SEC;
}

double const_memory(unsigned char *image, int width, int height) {
  clock_t start = clock();

  return (clock() - start) / (double) CLOCKS_PER_SEC;
}


void magic(const char* filename) {


  string target(filename);
  stringstream ss(target);
  while(getline(ss, target, '/'));

  vector<unsigned char> image; //the raw pixels
  unsigned int width, height;
  unsigned int error = lodepng::decode(image, width, height, filename, LCT_GREY);
  if(error) cout << "decoder error " << error << ": " << lodepng_error_text(error) << endl;

  unsigned char *data = (unsigned char *) malloc(width * height * sizeof (unsigned char));
  unsigned char *ans  = (unsigned char *) malloc(width * height * sizeof (unsigned char));

  for (int i = 0; i < image.size(); ++i)
    data[i] = image[i];

  char sobel_y[] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};
  // char sobel_x[] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
  // double sec_time = sequential(data, ans, width, height, sobel_x, 3);
  double sec_time = sequential(data, ans, width, height, sobel_y, 3);
  image = vector<unsigned char>(ans, ans +  width * height);
  error = lodepng::encode(string("seq_") + target, image, width, height, LCT_GREY);
  if(error) cout << "encoder error " << error << ": " << lodepng_error_text(error) << endl;

  double glm_time = global_memory(data, ans, width, height, sobel_y, 3);
  image = vector<unsigned char>(ans, ans +  width * height);
  error = lodepng::encode(string("par_") + target, image, width, height, LCT_GREY);
  if(error) cout << "encoder error " << error << ": " << lodepng_error_text(error) << endl;

  cout << image.size() << '\t' << sec_time << '\t' << glm_time  << endl;
  free(data);
  free(ans);
}

int main(int argc, char **argv) {
  if (argc > 1)
    magic(argv[1]);
  else
    magic("../images/cat1.png");
  return 0;
}

