#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <cstdlib>
#include <cassert>
#include <ctime>
#include <cmath>

#define CUDA_CALL(F)  if( (F) != hipSuccess ) \
{printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
    __FILE__,__LINE__); exit(-1);}
#define CUDA_CHECK()  if( (hipPeekAtLastError()) != hipSuccess ) \
{printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
    __FILE__,__LINE__-1); exit(-1);}

#include "../../utils.cu"

#define THPB 7

double _filter[]    = {0.006, 0.062, 0.242, 0.383, 0.242, 0.061, 0.006};
int filter_size = 7;

void convolution_seq(double *in, double *filter, double *out, int n, int f_size) {
  clock_t start = clock();
  f_size = f_size >> 1;
  for (int i = 0; i < n; ++i) {
    out[i] = 0;
    for (int j = -f_size; j <= f_size; ++j) {
      if (i + j >= 0 && i + j < n) {
        out[i] += in[i + j] * filter[j + f_size];
      }
    }
  }
  printf(" %.10lf ", (double)(clock() - start) / CLOCKS_PER_SEC);
}

__global__ void naive(double *in, double *filter, double *out, int n, int f_size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  double val = 0;
  int hs = f_size >> 1;
  if (idx < n) {
    for (int i = -hs; i <= hs; ++i) {
      if ((idx + i) >= 0 && (idx + i) < n)
        val += in[idx + i] * filter[i + hs];
    }
    out[idx] = val;
  }
}

__global__ void tiled(double *in, double *filter, double *out, int n, int f_size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int hs  =  f_size >> 1;
  int next = (blockIdx.x + 1) * blockDim.x;
  extern __shared__ double tile[];

  if (idx < n) {
    // tile[threadIdx.x + hs] = in[idx];
    out[threadIdx.x + hs] = idx;
  }


  if (idx < hs) {
    out[threadIdx.x] = hs + THPB + threadIdx.x;
  }

  if (threadIdx.x > 3)
    out[hs + THPB + threadIdx.x] = idx;
/*
 *  if (threadIdx.x < hs) {
 *    if (idx >= hs)
 *      // tile[threadIdx.x] = in[blockDim.x * blockIdx.x - hs + threadIdx.x];
 *      out[threadIdx.x] = blockDim.x * blockIdx.x - hs + threadIdx.x;
 *    else
 *      // tile[threadIdx.x] = -1;
 *      out[threadIdx.x] = -1;
 *
 *
 *  }
 *  if (threadIdx.x >= THPB - hs) {
 *    int cur = threadIdx.x - THPB - hs;
 *    if (next + cur < n) {
 *       // tile[cur + THPB + hs] = in[next + cur];
 *       out[cur + THPB + hs] = in[next + cur];
 *    }
 *  }
 *  __syncthreads();
 */

  /*
   *double val = 0;
   *for (int i = 0; i < f_size; ++i) {
   *  if (idx < n)
   *    val += tile[i + threadIdx.x] * filter[i];
   *}
   */

  // if (idx < n)
  //  out[idx] = val;

}

__global__ void const_me(double *in, double *filter, double *out, int n, int f_size) {

}

void convolution_par(double *in, double *filter, double *out, int n, int f_size) {
  clock_t start = clock();
  double *d_in, *d_filter, *d_out;
  CUDA_CALL(hipMalloc(&d_in, n * sizeof(double)));
  CUDA_CALL(hipMalloc(&d_filter, f_size * sizeof(double)));
  CUDA_CALL(hipMalloc(&d_out, n * sizeof(double)));

  CUDA_CALL(hipMemcpy(d_in, in, n * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_filter, filter, f_size * sizeof(double), hipMemcpyHostToDevice));

  dim3 dim_grid((n + THPB - 1) / THPB, 1, 1);
  dim3 dim_block(THPB, 1, 1);

  naive<<< dim_grid, dim_block >>> (d_in, d_filter, d_out, n, f_size);
  hipDeviceSynchronize();
  CUDA_CHECK();

  CUDA_CALL(hipMemcpy(out, d_out, n * sizeof(double), hipMemcpyDeviceToHost));

  CUDA_CALL(hipFree(d_in));
  CUDA_CALL(hipFree(d_filter));
  CUDA_CALL(hipFree(d_out));
  printf(" %.10lf ", (double)(clock() - start) / CLOCKS_PER_SEC);
}

void convolution_tiled(double *in, double *filter, double *out, int n, int f_size) {
  clock_t start = clock();
  double *d_in, *d_filter, *d_out;
  CUDA_CALL(hipMalloc(&d_in, n * sizeof(double)));
  CUDA_CALL(hipMalloc(&d_filter, f_size * sizeof(double)));
  CUDA_CALL(hipMalloc(&d_out, n * sizeof(double)));

  CUDA_CALL(hipMemcpy(d_in, in, n * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_filter, filter, f_size * sizeof(double), hipMemcpyHostToDevice));

  // Kernel cool stuff.
  dim3 dim_grid((n + THPB - 1) / THPB, 1, 1);
  dim3 dim_block(THPB, 1, 1);

  tiled<<< dim_grid, dim_block, (THPB + f_size) * sizeof (double) >>> (d_in, d_filter, d_out, n, f_size);
  hipDeviceSynchronize();
  CUDA_CHECK();

  CUDA_CALL(hipMemcpy(out, d_out, n * sizeof(double), hipMemcpyDeviceToHost));

  CUDA_CALL(hipFree(d_in));
  CUDA_CALL(hipFree(d_filter));
  CUDA_CALL(hipFree(d_out));
  printf(" %.10lf ", (double)(clock() - start) / CLOCKS_PER_SEC);
}

void convolution_const(double *in, double *filter, double *out, int n, int f_size) {
  clock_t start = clock();
  double *d_in, *d_filter, *d_out;
  CUDA_CALL(hipMalloc(&d_in, n * sizeof(double)));
  CUDA_CALL(hipMalloc(&d_filter, f_size * sizeof(double)));
  CUDA_CALL(hipMalloc(&d_out, n * sizeof(double)));

  CUDA_CALL(hipMemcpy(d_in, in, n * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_filter, filter, f_size * sizeof(double), hipMemcpyHostToDevice));

  // Kernel cool stuff.


  CUDA_CALL(hipMemcpy(out, d_out, n * sizeof(double), hipMemcpyDeviceToHost));

  CUDA_CALL(hipFree(d_in));
  CUDA_CALL(hipFree(d_filter));
  CUDA_CALL(hipFree(d_out));
  printf(" %.10lf ", (double)(clock() - start) / CLOCKS_PER_SEC);
}

void go_out(double *output_hos, double *output_dev, int n) {
#if 1
  puts("");
  for (int i = 0; i < n; ++i)
    printf("%.5lf ", output_hos[i]);
  puts("");
  for (int i = 0; i < n; ++i)
    printf("%.5lf ", output_dev[i]);
  puts("");
#endif
  exit(1);
}

int main() {
  srand(time(0));
  int lengths[] = {13, 1024, 1048576};
  int num_test  = 3;
  for (int tc = 0; tc < num_test; ++tc) {
    int n = lengths[tc];
    double *input  = new double[n];
    double *output_hos = new double[n];
    double *output_dev = new double[n];
    fill_random_vec(input, n);
    convolution_seq(input, _filter, output_hos, n, filter_size);
    convolution_par(input, _filter, output_dev, n, filter_size);
    if (!cmp_vect(output_hos, output_dev, n)) {
      fprintf(stderr, "Problem wiht pararallel (naive) convolution on test %d\n", tc);
      go_out(output_hos, output_dev, n);
    }

    convolution_tiled(input, _filter, output_dev, n, filter_size);
    if (!cmp_vect(output_hos, output_dev, n)) {
      fprintf(stderr, "Problem wiht parallel (tiled) convolution on test %d\n", tc);
      go_out(output_hos, output_dev, n);
    }

    // convolution_const(input, _filter, output_dev, n, filter_size);
    if (!cmp_vect(output_hos, output_dev, n)) {
      fprintf(stderr, "Problem wiht pararallel (constant) convolution on test %d\n", tc);
      exit(1);
    }
    delete [] input, output_dev, output_hos;
  }
  return 0;
}
