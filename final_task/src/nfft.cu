
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
using namespace std;

typedef long long int LL;
typedef pair<LL, LL> PLL;

/* The following vector of pairs contains pairs (prime, generator) where the prime has an Nth
 * root of unity for N being a power of two. The generator is a number g s.t g^(p-1)=1 (mod p)
 * but is different from 1 for all smaller powers */
PLL ROU[] = {make_pair(1224736769,330732430), make_pair(1711276033,927759239),
            make_pair(167772161,167489322), make_pair(469762049,343261969),
            make_pair(754974721,643797295), make_pair(1107296257,883865065)};

PLL ROU_2[] = {make_pair(1711276033LL, 1223522572LL), make_pair(1790967809LL, 1110378081LL)};

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

PLL ext_euclid(LL a, LL b) {
  if (b == 0)
    return make_pair(1,0);
  pair<LL,LL> rc = ext_euclid(b, a % b);
  return make_pair(rc.second, rc.first - (a / b) * rc.second);
}

//returns -1 if there is no unique modular inverse
LL mod_inv(LL x, LL modulo) {
  PLL p = ext_euclid(x, modulo);
  if ( (p.first * x + p.second * modulo) != 1 )
    return -1;
  return (p.first+modulo) % modulo;
}

// Computes ( a ^ exp ) % mod.
LL mod_pow(LL a, LL exp, LL mod) {
  LL ans = 1, base = a;
  while (exp > 0) {
    if (exp & 1)
      ans = (ans * base) % mod;
    base = (base * base) % mod;
    exp >>= 1;
  }
  return ans;
}


//Number theory fft. The size of a must be a power of 2
void ntfft(vector<LL> &a, int dir, const PLL &root_unity) {
  int n = a.size();
  LL prime = root_unity.first;
  LL basew = mod_pow(root_unity.second, (prime-1) / n, prime);
  if (dir < 0) basew = mod_inv(basew, prime);
  for (int m = n; m >= 2; m >>= 1) {
    int mh = m >> 1;
    LL w = 1;
    for (int i = 0; i < mh; i++) {
      for (int j = i; j < n; j += m) {
        int k = j + mh;
        LL x = (a[j] - a[k] + prime) % prime;
        a[j] = (a[j] + a[k]) % prime;
        a[k] = (w * x) % prime;
      }
      w = (w * basew) % prime;
    }
    basew = (basew * basew) % prime;
  }
  int i = 0;
  for (int j = 1; j < n - 1; j++) {
    for (int k = n >> 1; k > (i ^= k); k >>= 1);
    if (j < i) swap(a[i], a[j]);
  }
}

int bit_reverse(int x, int n) {
  int ans = 0;
  for (int i = 0; i < n; i++)
    if ((x >> i) & 1)
      ans |= ((1 << (n - i - 1)));
  return ans;
}

void bit_reverse_copy(LL *a, LL *A, int n, int size) {
  for (int i = 0; i < size; i++)
    A[bit_reverse(i, n)] = a[i];
}

void compute_powers(LL *powers, int ln, LL basew, LL prime){
  powers[0] = basew;
  for (int i = 1; i < ln; i++){
    powers[i] = (powers[i - 1] * powers[i - 1]) % prime;
  }
}

void fft(LL *a, LL *A, int dir, LL prime, LL basew, int size) {
  int ln = ceil(log2(float(size)));
  bit_reverse_copy(a, A, ln, size);
  LL *powers = (LL*) malloc (sizeof (LL) * ln);
  compute_powers(powers, ln, basew, prime);

  for (int s = 1; s <= ln; s++) {
    long long m = (1LL << s);
    LL wm = powers[ln - s];
    if (dir == -1)
      wm =  mod_inv(wm, prime);

    for (int k = 0; k < size; k += m) {
      LL w = 1, mh = m >> 1;
      for (int j = 0; j < mh; j++) {
        LL t = (w * A[k + j + mh]) % prime;
        LL u = A[k + j];
        A[k + j] = (u + t) % prime;
        A[k + j + mh] = (u - t + prime) % prime;
        w = (w * wm) % prime;
      }
    }
  }

  if (dir < 0) {
    LL in = mod_inv(size, prime);
    for (int i = 0; i < size; i++)
      A[i] = (A[i] * in) % prime;
  }
}

__device__ void d_ext_euclid(LL a, LL b, LL &x, LL &y, LL &g) {
  x = 0, y = 1, g = b;
  LL m, n, q, r;
  for (LL u = 1, v = 0; a != 0; g = a, a = r) {
    q = g / a, r = g % a;
    m = x - u * q, n = y - v * q;
    x = u, y = v, u = m, v = n;
  }
}

__device__ LL d_mod_inv(LL n, LL m) {
  LL x, y, gcd;
  d_ext_euclid(n, m, x, y, gcd);
  if (gcd != 1)
    return 0;
  return (x + m) % m;
}


__global__ void fft_kernel (LL *A, int dir, LL prime, int ln, LL *powers, int size){
  int pos = threadIdx.x + blockDim.x * blockIdx.x;
  for (int s = 1; s <= ln; s++){
    LL m = (1LL << s);
    LL wm = powers[ln -s];
    int k = pos * m;
    if (dir == -1)
      wm = d_mod_inv (wm, prime);
    if (k >= size)
      return;
    else{
      LL w = 1;
      LL mh = m >> 1;
      for (int j = 0; j < mh; j++){
        LL t = (w * A[k + j + mh]) % prime;
        LL u = A[k + j];
        A[k + j] = (u + t) % prime;
        A[k + j + mh] = (u - t + prime) % prime;
        w = (w * wm) % prime;
      }
    }
    __syncthreads();
  }

  if (dir < 0) {
    LL in = d_mod_inv(size, prime);
    for (int i = 0; i < size; i++)
      A[i] = (A[i] * in) % prime;
  }
}

void fft_con(LL *a, LL *A, int dir, LL prime, LL basew, int size){
  int ln = ceil(log2(float(size)));
  bit_reverse_copy(a, A, ln, size);
  LL *powers = (LL*) malloc (sizeof (LL) * ln);
  compute_powers(powers, ln, basew, prime);

  LL *d_A, *d_powers;
  hipMalloc(&d_A, size * sizeof(LL));
  hipMalloc(&d_powers, ln * sizeof(LL));

  hipMemcpy (d_A, A, size * sizeof (LL), hipMemcpyHostToDevice);
  hipMemcpy (d_powers, powers, ln * sizeof (LL), hipMemcpyHostToDevice);

  dim3 dimGrid(ceil(float(size / 1024.0)), 1, 1);
  dim3 dimBlock(1024, 1, 1);

  fft_kernel<<<dimGrid, dimBlock>>> (d_A, dir, prime, ln, d_powers, size);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  hipMemcpy (A, d_A, size * sizeof (LL), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_powers);
  free(powers);

}

bool cmp_vectors (LL *A, LL *B, int size){
  for (int i = 0; i < size; i++){
    if (A[i] != B[i]){
      cout << A[i] << " " << B[i] << " i: " << i << endl;
      return false;
    }
  }
  return true;
}

int main(){
  LL prime = ROU_2[0].first;
  LL basew = ROU_2[0].second;
  int size = 1024;
  LL *a = (LL*) malloc( sizeof (LL) * size);
  LL *A = (LL*) malloc( sizeof (LL) * size);
  LL *B = (LL*) malloc( sizeof (LL) * size);
  for (int i = 0; i < size; i++){
    a[i] = i;
  }

  fft(a, A, 1, prime, basew, size);
  fft_con(a, B, 1, prime, basew, size);

  if (!cmp_vectors (A, B, size))
    cout << "nay :(" << endl;
  else
    cout << "yay :)" << endl;

  for (int i = 0; i < size; i++)
    cout << B[i] << endl;

  free(a);
  free(A);
  free(B);
  return 0;
}
